// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -out-root %T/cos %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/cos/cos.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/cos/cos.dp.cpp -o %T/cos/cos.dp.o %}

// clang-format off
#include <hip/hip_runtime.h>
#include <cstdint>

__global__ void cos() {
  float f32;

  // CHECK: f32 = sycl::cos(1.0f);
  asm("cos.approx.f32 %0, %1;" : "=f"(f32) : "f"(1.0f));
  
  // CHECK: f32 = sycl::cos(1.0f);
  asm("cos.approx.ftz.f32 %0, %1;" : "=f"(f32) : "f"(1.0f));
}

// clang-format on
