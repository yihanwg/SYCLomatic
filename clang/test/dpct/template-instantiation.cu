// RUN: dpct --format-range=none -out-root %T/template-instantiation %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -std=c++14
// RUN: FileCheck %s --match-full-lines --input-file %T/template-instantiation/template-instantiation.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/template-instantiation/template-instantiation.dp.cpp -o %T/template-instantiation/template-instantiation.dp.o %}


#include <hip/hip_runtime.h>
#include <vector>

template<class T>
T &host_instantiation(T &a) { return a; }

// CHECK: template const std::vector<sycl::float2> &host_instantiation(std::vector<sycl::float2> const &);
template const std::vector<float2> &host_instantiation(std::vector<float2> const &);

// CHECK: void kernel(int *, const sycl::nd_item<3> &item_ct1, T *a);
template<class T>
__global__ void kernel(int *);

// CHECK: template void kernel<sycl::float2>(int *, const sycl::nd_item<3> &item_ct1,
// CHECK-NEXT: sycl::float2 *a);
template __global__ void kernel<float2>(int *);

// CHECK: template void kernel<int>(int *, const sycl::nd_item<3> &item_ct1, int *a);
template __global__ void kernel<int>(int *);

template<class T>
// CHECK: void kernel1(T *, const sycl::nd_item<3> &item_ct1, T *a) {
__global__ void kernel1(T *) {
  __shared__ T a[10];
  int b = blockDim.x;
}

// CHECK: template void kernel1(sycl::char4 *, const sycl::nd_item<3> &item_ct1, sycl::char4 *a);
template __global__ void kernel1(char4 *);

// CHECK: template void kernel1<int>(int *, const sycl::nd_item<3> &item_ct1, int *a);
template __global__ void kernel1<int>(int *);

// CHECK: void kernel2(T1 *, T2 *, const sycl::nd_item<3> &item_ct1, T1 *a1, T2 *a2) {
template<class T1, class T2>
__global__ void kernel2(T1 *, T2 *) {
  __shared__ T1 a1[10];
  __shared__ T2 a2[10];
  int b = blockDim.x;
}

// CHECK: template void kernel2(sycl::char4 *, int *, const sycl::nd_item<3> &item_ct1,
// CHECK-NEXT: sycl::char4 *a1, int *a2);
template __global__ void kernel2(char4 *, int *);

// CHECK: template void kernel2<int>(int *, sycl::float2 *, const sycl::nd_item<3> &item_ct1,
// CHECK-NEXT:  int *a1, sycl::float2 *a2);
template __global__ void kernel2<int>(int *, float2 *);

// CHECK: template void kernel2<sycl::float2, sycl::char4>(sycl::float2 *, sycl::char4 *,
// CHECK-NEXT:   const sycl::nd_item<3> &item_ct1,
// CHECK-NEXT:   sycl::float2 *a1, sycl::char4 *a2);
template __global__ void kernel2<float2, char4>(float2 *, char4 *);

template<unsigned S, class T>
// CHECK: void kernel3(T *, const sycl::nd_item<3> &item_ct1, T *a) {
__global__ void kernel3(T *) {
  __shared__ T a[S];
  int b = blockDim.x;
}
// CHECK: template void kernel3<20>(int *, const sycl::nd_item<3> &item_ct1, int *a);
template __global__ void kernel3<20>(int *);

template <typename T> void func_2_same_pram(T a, T b) {}

template <typename T> T func_same_return(T a) { return a; }

int main() {
    int *d;
    float2 *d1;
    int4 *d2;

// CHECK:      q_ct1.submit(
// CHECK-NEXT:   [&](sycl::handler &cgh) {
// CHECK-NEXT:     sycl::local_accessor<sycl::float2, 1> a_acc_ct1(sycl::range<1>(10), cgh);
// CHECK-EMPTY:
// CHECK-NEXT:     cgh.parallel_for(
// CHECK-NEXT:       sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
// CHECK-NEXT:       [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:         kernel<sycl::float2>(d, item_ct1, a_acc_ct1.template get_multi_ptr<sycl::access::decorated::no>().get());
// CHECK-NEXT:     });
// CHECK-NEXT: });
    kernel<float2><<<1,1>>>(d);

// CHECK:      q_ct1.submit(
// CHECK-NEXT:   [&](sycl::handler &cgh) {
// CHECK-NEXT:     sycl::local_accessor<sycl::int4, 1> a_acc_ct1(sycl::range<1>(10), cgh);
// CHECK-EMPTY:
// CHECK-NEXT:     cgh.parallel_for(
// CHECK-NEXT:       sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
// CHECK-NEXT:       [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:         kernel<sycl::int4>(d, item_ct1, a_acc_ct1.template get_multi_ptr<sycl::access::decorated::no>().get());
// CHECK-NEXT:     });
// CHECK-NEXT: });
    kernel<int4><<<1,1>>>(d);

// CHECK:      q_ct1.submit(
// CHECK-NEXT:   [&](sycl::handler &cgh) {
// CHECK-NEXT:     sycl::local_accessor<int, 1> a_acc_ct1(sycl::range<1>(10), cgh);
// CHECK-EMPTY:
// CHECK-NEXT:     cgh.parallel_for(
// CHECK-NEXT:       sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
// CHECK-NEXT:       [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:         kernel1(d, item_ct1, (int *)a_acc_ct1.template get_multi_ptr<sycl::access::decorated::no>().get());
// CHECK-NEXT:     });
// CHECK-NEXT: });
    kernel1<<<1,1>>>(d);

// CHECK:      q_ct1.submit(
// CHECK-NEXT:   [&](sycl::handler &cgh) {
// CHECK-NEXT:     sycl::local_accessor<int, 1> a1_acc_ct1(sycl::range<1>(10), cgh);
// CHECK-NEXT:     sycl::local_accessor<sycl::float2, 1> a2_acc_ct1(sycl::range<1>(10), cgh);
// CHECK-EMPTY:
// CHECK-NEXT:     cgh.parallel_for(
// CHECK-NEXT:       sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
// CHECK-NEXT:       [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:         kernel2<int>(d, d1, item_ct1, a1_acc_ct1.template get_multi_ptr<sycl::access::decorated::no>().get(), (sycl::float2 *)a2_acc_ct1.template get_multi_ptr<sycl::access::decorated::no>().get());
// CHECK-NEXT:     });
// CHECK-NEXT: });
    kernel2<int><<<1,1>>>(d, d1);

// CHECK:      q_ct1.submit(
// CHECK-NEXT:   [&](sycl::handler &cgh) {
// CHECK-NEXT:     sycl::local_accessor<int, 1> a_acc_ct1(sycl::range<1>(20), cgh);
// CHECK-EMPTY:
// CHECK-NEXT:     cgh.parallel_for(
// CHECK-NEXT:       sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
// CHECK-NEXT:       [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:         kernel3<20>(d, item_ct1, (int *)a_acc_ct1.template get_multi_ptr<sycl::access::decorated::no>().get());
// CHECK-NEXT:     });
// CHECK-NEXT: });
    kernel3<20><<<1,1>>>(d);

    unsigned u;
    dim3 dim;
    // CHECK: func_2_same_pram(u, dim.y);
    func_2_same_pram(u, dim.y);
    // CHECK: func_2_same_pram(u, dim.y + 1);
    func_2_same_pram(u, dim.y + 1);
    // CHECK: func_2_same_pram(u, func_same_return(dim.y));
    func_2_same_pram(u, func_same_return(dim.y));
}

// CHECK: void kernel(int *, const sycl::nd_item<3> &item_ct1, T *a) {
template<class T>
__global__ void kernel(int *) {
  __shared__ T a[10];
  int b = blockDim.x;
}

template <typename T> void f() {}
template <typename T> class CCCCCCCCCCC {};
// CHECK: template void f<CCCCCCCCCCC<CCCCCCCCCCC<CCCCCCCCCCC<sycl::int3>>>>();
template void f<CCCCCCCCCCC<CCCCCCCCCCC<CCCCCCCCCCC<int3>>>>();
// CHECK: template void f<CCCCCCCCCCC<CCCCCCCCCCC<sycl::int3>>>();
template void f<CCCCCCCCCCC<CCCCCCCCCCC<int3>>>();
// CHECK: template void f<CCCCCCCCCCC<sycl::int3>>();
template void f<CCCCCCCCCCC<int3>>();
// CHECK: template void f<sycl::int3>();
template void f<int3>();
