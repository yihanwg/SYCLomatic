// Option: --use-dpcpp-extensions=intel_device_math

#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"
#include "hip/hip_fp16.h"

__global__ void test(__half2 h1, __half2 h2, __hip_bfloat162 b1,
                     __hip_bfloat162 b2) {
  // Start
  __hmax2_nan(h1 /*__half2*/, h2 /*__half2*/);
  __hmax2_nan(b1 /*__nv_bfloat162*/, b2 /*__nv_bfloat162*/);
  // End
}
