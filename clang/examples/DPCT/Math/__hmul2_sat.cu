// Option: --use-dpcpp-extensions=intel_device_math

#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"
#include "hip/hip_fp16.h"

__global__ void test(__half2 h1, __half2 h2, __hip_bfloat162 b1,
                     __hip_bfloat162 b2) {
  // Start
  __hmul2_sat(h1 /*__half2*/, h2 /*__half2*/);
  __hmul2_sat(b1 /*__nv_bfloat162*/, b2 /*__nv_bfloat162*/);
  // End
}
