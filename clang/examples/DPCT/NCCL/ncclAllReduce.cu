
#include <hip/hip_runtime.h>
#include <nccl.h>

void test(const void *sendbuff, void *recvbuff, size_t count,
          ncclDataType_t datatype, ncclRedOp_t op, ncclComm_t comm,
          hipStream_t stream) {
  // Start
  ncclAllReduce(sendbuff /*void **/, recvbuff /*void **/, count /*size_t*/,
                datatype /*ncclDataType_t*/, op /*ncclRedOp_t*/,
                comm /*ncclComm_t*/, stream /*cudaStream_t*/);
  // End
}