
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const float *x, int incx, float *y,
          int incy) {
  // Start
  hipblasScopy(handle /*cublasHandle_t*/, n /*int*/, x /*const float **/,
              incx /*int*/, y /*float **/, incy /*int*/);
  // End
}
